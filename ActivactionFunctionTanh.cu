#include "hip/hip_runtime.h"
/*
 * ActivactionFunctionTanh.cu
 *
 *  Created on: Jan 30, 2012
 *      Author: wchan
 */

/*
 * Note: This file is needed because the nvcc compiler doesn't support C++0x
 *
 * When the nvcc compiler support comes, this can be integrated back in the .hpp file
 */

#include "ActivationFunctionTanh.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/transform.h>

struct tanh_func {
  __device__
  double operator()(double x) {
    return tanh(x);
  }
};

struct dtanh_func {
  __device__
  double operator()(double y) {
    return 1.0 - y * y;
  }
};

void ActivationFunctionTanh::f_cuda(double* x, size_t size) {
  thrust::device_ptr<double> ptr(x);

  thrust::transform(ptr, ptr + size, ptr, tanh_func());
}

void ActivationFunctionTanh::d_cuda(double* y, size_t size, double* d) {
  thrust::device_ptr<double> ptr0(y);
  thrust::device_ptr<double> ptr1(d);

  thrust::transform(ptr0, ptr0 + size, ptr1, dtanh_func());
}

